#include "hip/hip_runtime.h"
// g++ readpixel.cpp -o readpixel -lGL -lglut
#define GL_GLEXT_PROTOTYPES
#include <GL/gl.h>
#include <GL/glext.h>
#include <GL/glut.h>
#include <stdio.h>
#include <cuda_gl_interop.h>

int mx = 0, my = 0;
int WIDTH=800, HEIGHT=600;
GLuint pbo=0;
char * bits = NULL;
int usepbo = 1;
struct hipGraphicsResource *cuda_pbo_dest_resource;

// Some dummy kernel to prevent optimizations
__global__ void kernel(char* in)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    in[index] = in[index]-1;
}

#define cutilSafeCall(err)  __cudaSafeCall(err,__FILE__,__LINE__)
inline void __cudaSafeCall(hipError_t err,
                           const char *file, const int line){
  if(hipSuccess != err) {
    printf("%s(%i) : cutilSafeCall() Runtime API error : %s.\n",
           file, line, hipGetErrorString(err) );
    exit(-1);
  }
}

void display()
{
    glClearColor( 0, 0, 0, 1 );
    glClear( GL_COLOR_BUFFER_BIT );

    glMatrixMode( GL_PROJECTION );
    glLoadIdentity();
    glOrtho( -10, 10, -10, 10, -1, 1 );

    glMatrixMode( GL_MODELVIEW );
    glLoadIdentity();

    glScalef( 5, 5, 5 );
    glBegin( GL_TRIANGLES );
    glColor3ub( 255, 0, 0 );
    glVertex2f( -1, -1 );
    glColor3ub( 0, 255, 0 );
    glVertex2f( 1, -1 );
    glColor3ub( 0, 0, 255 );
    glVertex2f( 0, 1 );
    glEnd();

    // 4 bytes per pixel (RGBA), 1x1 bitmap
    if(!usepbo){
        unsigned char pixels[ 1 * 1 * 4 ] = { 0 };
        glReadPixels( mx, my, 1, 1, GL_RGBA, GL_UNSIGNED_BYTE, pixels );
        printf("r: %d\n", pixels[ 0 ] );
        printf("g: %d\n", pixels[ 1 ] );
        printf("b: %d\n", pixels[ 2 ] );
        printf("a: %d\n", pixels[ 3 ] );
    }else{
        glReadBuffer(GL_FRONT);
        glBindBuffer(GL_PIXEL_PACK_BUFFER, pbo);
        glReadPixels(0, 0, WIDTH, HEIGHT, GL_BGRA, GL_UNSIGNED_BYTE, 0);
        //unsigned char* pboBits = NULL;
        //pboBits = (unsigned char *)glMapBuffer(GL_PIXEL_PACK_BUFFER, GL_READ_ONLY);
        //if(!pboBits){printf("Could not map pboBits\n"); return;}
        //printf("r: %d\n", pboBits[(my*WIDTH+ mx)*4 + 0 ] );
        //printf("g: %d\n", pboBits[(my*WIDTH+ mx)*4 + 1 ] );
        //printf("b: %d\n", pboBits[(my*WIDTH+ mx)*4 + 2 ] );
        //printf("a: %d\n", pboBits[(my*WIDTH+ mx)*4 + 3 ] );
        //glUnmapBuffer(GL_PIXEL_PACK_BUFFER_EXT);

        cutilSafeCall(hipGraphicsMapResources(1, &cuda_pbo_dest_resource));
        size_t size = 0;
        void* device_ptr = NULL;
        cutilSafeCall( hipGraphicsResourceGetMappedPointer(&device_ptr, &size, cuda_pbo_dest_resource) );
        glBindBuffer(GL_PIXEL_PACK_BUFFER_EXT, 0);

        kernel<<<size/512,512>>>((char*)device_ptr);
        hipDeviceSynchronize();
        cutilSafeCall( hipGraphicsUnmapResources(1, &cuda_pbo_dest_resource) );

    }

    glutSwapBuffers();
}

void mouse( int x, int y )
{
    mx = x;
    my = glutGet( GLUT_WINDOW_HEIGHT ) - y;
    glutPostRedisplay();
}

void init(){
        if(!pbo) {
             glGenBuffers(1, &pbo);
             glBindBuffer(GL_PIXEL_PACK_BUFFER_EXT, pbo);
             glBufferData(GL_PIXEL_PACK_BUFFER_EXT, WIDTH * HEIGHT * 4, NULL,GL_STREAM_READ);
             int size = 0;
             glGetBufferParameteriv(GL_PIXEL_PACK_BUFFER_EXT, GL_BUFFER_SIZE, &size);
             if(size != WIDTH * HEIGHT*4){
                 printf("Could not set PBO size\n");
                 return;
             }
             //bits = (char *)malloc(WIDTH*HEIGHT*4);
             cutilSafeCall( hipGraphicsGLRegisterBuffer(&cuda_pbo_dest_resource, pbo, cudaGraphicsMapFlagsNone) );
        }

}

void end(){
    // Unmap and unregister the graphics resource
    cutilSafeCall( hipGraphicsUnregisterResource(cuda_pbo_dest_resource) );
}

int main( int argc, char **argv )
{
    glutInit( &argc, argv );
    glutInitDisplayMode( GLUT_RGBA | GLUT_DOUBLE );
    glutInitWindowSize(WIDTH, HEIGHT);
    glutCreateWindow( "glReadPixels() example" );
    glutDisplayFunc( display );
    glutPassiveMotionFunc( mouse );
    init();
    glutMainLoop();
    end();
    return 0;
}
