#include "hip/hip_runtime.h"
// g++ readpixel.cpp -o readpixel -lGL -lglut

#define GL_GLEXT_PROTOTYPES
#include <GL/gl.h>
#include <GL/glext.h>
#include <GL/glut.h>
#include <stdio.h>
#include <cuda_gl_interop.h>

#define WIDTH 800
#define HEIGHT 600

int mx = 0, my = 0;
GLuint pbo=0;
char * bits = NULL;
int usepbo = 1;
struct hipGraphicsResource *cuda_pbo_dest_resource;

#define cutilSafeCall(err)  __cudaSafeCall(err,__FILE__,__LINE__)
inline void __cudaSafeCall(hipError_t err,
                           const char *file, const int line){
  if(hipSuccess != err) {
    printf("%s(%i) : cutilSafeCall() Runtime API error : %s.\n",
           file, line, hipGetErrorString(err) );
    exit(-1);
  }
}

__global__ void rgb2yuv420Kernel(unsigned char* rgbImg, unsigned char* yuv420){
    unsigned char r, g, b;
    unsigned char y, cb, cr;
 
    int index,i,j;
    index = threadIdx.x + blockIdx.x * blockDim.x;
    i = index/WIDTH;
    j = index%WIDTH;
    
    r = rgbImg[index*3 + 0];
    g = rgbImg[index*3 + 1];
    b = rgbImg[index*3 + 2];
     
    y  = (unsigned char)( 0.299*r + 0.587*g +  0.114*b);
    cb = (unsigned char)(-0.147*r - 0.289*g +  0.463*b + 128);
    cr = (unsigned char)( 0.615*r - 0.515*g - 0.1*b + 128);
     
    rgbImg[index*3 + 0] = y;
    rgbImg[index*3 + 1] = cb;
    rgbImg[index*3 + 2] = cr;

    yuv420[index + 0] = y;
    if(i%2 == 0 && j%2 == 0){
        yuv420[(i/2)*(WIDTH/2) + (j/2) + WIDTH*HEIGHT] = cb;
        yuv420[(i/2)*(WIDTH/2) + (j/2) + WIDTH*HEIGHT*5/4] = cr;
    }
}

void nv_encoding(){
    m_cuContext->SetCurrent(); // Make Clients context current
    MapInputResource(id); //Map the CudaInputResource
    NV_ENC_PIC_PARAMS temp;
    memset(&temp, 0, sizeof(NV_ENC_PIC_PARAMS));
    temp.version = NV_ENC_PIC_PARAMS_VER;
    unsigned int currentBufferAndEvent = m_counter % m_registeredEvents.size(); //Counter is inc'ed in every Frame
    temp.bufferFmt = m_currentlyMappedInputBuffer.mappedBufferFmt;
    temp.inputBuffer = m_currentlyMappedInputBuffer.mappedResource; //got set by MapInputResource
    temp.completionEvent = m_registeredEvents[currentBufferAndEvent];
    temp.outputBitstream = m_registeredOutputBuffers[currentBufferAndEvent];
    temp.inputWidth = m_width;
    temp.inputHeight = m_height;
    temp.inputPitch = m_width;
    temp.inputTimeStamp = m_counter;
    temp.pictureStruct = NV_ENC_PIC_STRUCT_FRAME; // According to samples
    temp.qpDeltaMap = NULL;
    temp.qpDeltaMapSize = 0;

    EventWithId latestEvent(currentBufferAndEvent,
                  m_registeredEvents[currentBufferAndEvent]);
    PushBackEncodeEvent(latestEvent); // Store the Event with its ID in a Queue

    CheckApiError(m_apiFunctions.nvEncEncodePicture(m_Encoder, &temp));
    m_counter++;
    UnmapInputResource(id); // Unmap
}

void display()
{
    glClearColor( 0, 0, 0, 1 );
    glClear( GL_COLOR_BUFFER_BIT );

    glMatrixMode( GL_PROJECTION );
    glLoadIdentity();
    glOrtho( -10, 10, -10, 10, -1, 1 );

    glMatrixMode( GL_MODELVIEW );
    glLoadIdentity();

    glScalef( 5, 5, 5 );
    glBegin( GL_TRIANGLES );
    glColor3ub( 255, 0, 0 );
    glVertex2f( -1, -1 );
    glColor3ub( 0, 255, 0 );
    glVertex2f( 1, -1 );
    glColor3ub( 0, 0, 255 );
    glVertex2f( 0, 1 );
    glEnd();

    // 4 bytes per pixel (RGBA), 1x1 bitmap
    if(!usepbo){
        unsigned char pixels[ 1 * 1 * 3 ] = { 0 };
        glReadPixels( mx, my, 1, 1, GL_RGB, GL_UNSIGNED_BYTE, pixels );
        printf("r: %d\n", pixels[ 0 ] );
        printf("g: %d\n", pixels[ 1 ] );
        printf("b: %d\n", pixels[ 2 ] );
    }else{
        glReadBuffer(GL_FRONT);
        glBindBuffer(GL_PIXEL_PACK_BUFFER, pbo);
        glReadPixels(0, 0, WIDTH, HEIGHT, GL_RGB, GL_UNSIGNED_BYTE, 0);
        unsigned char* pboBits = NULL;
        pboBits = (unsigned char *)glMapBuffer(GL_PIXEL_PACK_BUFFER, GL_READ_ONLY);
        if(!pboBits){printf("Could not map pboBits\n"); return;}
        printf("r: %d\n", pboBits[(my*WIDTH+ mx)*3 + 0 ] );
        printf("g: %d\n", pboBits[(my*WIDTH+ mx)*3 + 1 ] );
        printf("b: %d\n", pboBits[(my*WIDTH+ mx)*3 + 2 ] );
        glUnmapBuffer(GL_PIXEL_PACK_BUFFER_EXT);

        cutilSafeCall(hipGraphicsMapResources(1, &cuda_pbo_dest_resource));
        size_t size = 0;
        void* device_ptr = NULL;
        cutilSafeCall( hipGraphicsResourceGetMappedPointer(&device_ptr, &size, cuda_pbo_dest_resource) );
        glBindBuffer(GL_PIXEL_PACK_BUFFER_EXT, 0);

        unsigned char* d_yuvOut;
        hipMalloc((void**)&d_yuvOut, WIDTH*HEIGHT*3/2);
        rgb2yuv420Kernel<<<WIDTH*HEIGHT/256,256>>>((unsigned char*)device_ptr, d_yuvOut);
        hipDeviceSynchronize();
        cutilSafeCall( hipGraphicsUnmapResources(1, &cuda_pbo_dest_resource) );

        
        glBindBuffer(GL_PIXEL_PACK_BUFFER, pbo);
        pboBits = NULL;
        pboBits = (unsigned char *)glMapBuffer(GL_PIXEL_PACK_BUFFER, GL_READ_ONLY);
        if(!pboBits){printf("Could not map pboBits\n"); return;}
        printf("y: %d\n", pboBits[(my*WIDTH+ mx)*3 + 0 ] );
        printf("Cb: %d\n", pboBits[(my*WIDTH+ mx)*3 + 1 ] );
        printf("Cr: %d\n", pboBits[(my*WIDTH+ mx)*3 + 2 ] );
        glUnmapBuffer(GL_PIXEL_PACK_BUFFER_EXT);
        glBindBuffer(GL_PIXEL_PACK_BUFFER_EXT, 0);

    }

    glutSwapBuffers();
}

void mouse( int x, int y )
{
    mx = x;
    my = glutGet( GLUT_WINDOW_HEIGHT ) - y;
    glutPostRedisplay();
}

void nvenc_init(){
    InitParams* ip = new InitParams();
    m_initParams = ip;
    memset(ip, 0, sizeof(InitParams));
    ip->version = NV_ENC_INITIALIZE_PARAMS_VER;
    ip->encodeGUID = m_encoderGuid;  //Used Codec
    ip->encodeWidth = width; // Frame Width
    ip->encodeHeight = height; // Frame Height
    ip->maxEncodeWidth = 0; // Zero means no dynamic res changes
    ip->maxEncodeHeight = 0; 
    ip->darWidth = width; // Aspect Ratio
    ip->darHeight = height; 
    ip->frameRateNum = 60; // 60 fps
    ip->frameRateDen = 1; 
    ip->reportSliceOffsets = 0; // According to programming guide
    ip->enableSubFrameWrite = 0;
    ip->presetGUID = m_presetGuid; // Used Preset for Encoder Config

    NV_ENC_PRESET_CONFIG presetCfg; // Load the Preset Config
    memset(&presetCfg, 0, sizeof(NV_ENC_PRESET_CONFIG));
    presetCfg.version = NV_ENC_PRESET_CONFIG_VER;
    presetCfg.presetCfg.version = NV_ENC_CONFIG_VER;
    CheckApiError(m_apiFunctions.nvEncGetEncodePresetConfig(m_Encoder,
                             m_encoderGuid, m_presetGuid, &presetCfg));
    memcpy(&m_encodingConfig, &presetCfg.presetCfg, sizeof(NV_ENC_CONFIG));
    // And add information about Bitrate etc
    m_encodingConfig.rcParams.averageBitRate = 500000;
    m_encodingConfig.rcParams.maxBitRate = 600000;
    m_encodingConfig.rcParams.rateControlMode = NV_ENC_PARAMS_RC_MODE::NV_ENC_PARAMS_RC_CBR;
    ip->encodeConfig = &m_encodingConfig;
    ip->enableEncodeAsync = 1; // Async Encoding
    ip->enablePTD = 1; // Encoder handles picture ordering

    // Registration of CudaResource
    m_cuContext->SetCurrent(); // Make the clients cuCtx current
    NV_ENC_REGISTER_RESOURCE res;
    memset(&res, 0, sizeof(NV_ENC_REGISTER_RESOURCE));
    NV_ENC_REGISTERED_PTR resPtr; // handle to the cuda resource for future use
    res.bufferFormat = m_inputFormat; // Format is ARGB
    res.height = m_height;
    res.width = m_width;
    // NOTE: I've set the pitch to the width of the frame, because the resource is a non-pitched 
    //hipArray. Is this correct? Pitch = 0 would produce no output.
    res.pitch = pitch; 
    res.resourceToRegister = (void*) (uintptr_t) resourceToRegister; //CUdevptr to resource
    res.resourceType = NV_ENC_INPUT_RESOURCE_TYPE::NV_ENC_INPUT_RESOURCE_TYPE_CUDADEVICEPTR;
    res.version = NV_ENC_REGISTER_RESOURCE_VER;
    CheckApiError(m_apiFunctions.nvEncRegisterResource(m_Encoder, &res));
    m_registeredInputResources.push_back(res.registeredResource);
}

void init(){
        if(!pbo) {
             glGenBuffers(1, &pbo);
             glBindBuffer(GL_PIXEL_PACK_BUFFER_EXT, pbo);
             glBufferData(GL_PIXEL_PACK_BUFFER_EXT, WIDTH * HEIGHT * 4, NULL,GL_STREAM_READ);
             int size = 0;
             glGetBufferParameteriv(GL_PIXEL_PACK_BUFFER_EXT, GL_BUFFER_SIZE, &size);
             if(size != WIDTH * HEIGHT*4){
                 printf("Could not set PBO size\n");
                 return;
             }
             //bits = (char *)malloc(WIDTH*HEIGHT*4);
             cutilSafeCall( hipGraphicsGLRegisterBuffer(&cuda_pbo_dest_resource, pbo, cudaGraphicsMapFlagsNone) );
        }
        nvenc_init();
}

void end(){
    // Unmap and unregister the graphics resource
    cutilSafeCall( hipGraphicsUnregisterResource(cuda_pbo_dest_resource) );
}

int main( int argc, char **argv )
{
    glutInit( &argc, argv );
    glutInitDisplayMode( GLUT_RGBA | GLUT_DOUBLE );
    glutInitWindowSize(WIDTH, HEIGHT);
    glutCreateWindow( "glReadPixels() example" );
    glutDisplayFunc( display );
    glutPassiveMotionFunc( mouse );
    init();
    glutMainLoop();
    end();
    return 0;
}
