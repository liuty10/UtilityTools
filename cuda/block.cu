#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <unistd.h>

__global__ void add(int *a, int *b, int *c){
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#define N 512
int random_ints(int *p, int n){
	int i;
	for(i=0;i<n;i++)
		*p++ = rand();
	return 0;
}


int main(){
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N*sizeof(int);

	hipSetDevice(1);
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);
	random_ints(a,N);
	random_ints(b,N);
	a[N-1] = 100;
	b[N-1] = 200;

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	add<<<N,1>>>(d_a, d_b, d_c);
	hipDeviceSynchronize();
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	printf("result = %d\n",c[N-1]);

	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
