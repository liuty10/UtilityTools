
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void add(int *a, int *b, int *c){
    *c = *a + *b;
}

int main(){
    int a, b, c;
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, sizeof(int));
    hipMalloc((void**)&d_b, sizeof(int));
    hipMalloc((void**)&d_c, sizeof(int));
    a = 3;
    b = 8;
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
    //add<<<1,1>>>(d_a, d_b, d_c);
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    printf("result =%d\n",c);
    return 0;
}
