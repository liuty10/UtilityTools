#include "hip/hip_runtime.h"

/*
 * Skeleton function for Parallel Computing Class, 
 * Assignment: K-Means Algorithm (CUDA)
 *
 * To students: You should finish the implementation of k_means algorithm.
 *              You should add device functions/kernels to perform k_means on 
 *              GPU. The "k_means" function in this file is just an interface
 *              for passing in basic parameters needed.. You need to add GPU 
 *              kernels and launch them in the "k_means" function.
 *
 *              Note that the "k_means" function has two input parameters for
 *              block count and thread count per block. Please use these two
 *              parameters when launching your kernels.
 * 
 * Author:
 *     Tianyi Liu <tianyi.liu@utsa.edu>
 */
#include <stdio.h>
#include <float.h>
#include <math.h>

#include "k_means.h"
__global__ void assignClusters(struct point *d_p, struct point *d_u, int *d_c, int *d_m, int *d_k, int block, int threads){
      int i,j;
      double dist;
      int taskPerWorker = (*d_m)/(block*threads);
      int lastTask = (*d_m)%(block*threads);
      if(lastTask!=0) taskPerWorker+=1;
      int index = blockIdx.x * blockDim.x + threadIdx.x;//number of thread
      if(index >= (*d_m)) return;
      for(j=index*taskPerWorker; j<(index+1)*taskPerWorker && j<(*d_m); j++){
          double min_dist = 1000000000000.0;
          for(i=0; i<(*d_k); i++){
                  dist = (d_p[j].x-d_u[i].x)*(d_p[j].x-d_u[i].x)+
                         (d_p[j].y-d_u[i].y)*(d_p[j].y-d_u[i].y);
                  if(dist < min_dist){
                          min_dist = dist;
                          d_c[j] = i;
                  }
          }
      }
}

void computeNewCenter(struct point *p, struct point *u, int* c, int m, int k){
      int i,j;
      for(j=0; j<k; j++){
              double sum_x = 0.0;
              double sum_y = 0.0;
              int cluster_size = 0;
              for(i=0; i<m; i++){
                      if(c[i] == j){
                              sum_x += p[i].x;
                              sum_y += p[i].y;
                              cluster_size++;
                      }
              }
              if(cluster_size > 0){
                      u[j].x = sum_x/cluster_size;
                      u[j].y = sum_y/cluster_size;
              }else{
                      u[j] = random_center(p);
              }
      }
}
/*
 * k_means: k_means clustering algorithm implementation.
 *
 * Input parameters:
 *     struct point p[]: array of data points
 *     int m           : number of data points in p[]
 *     int k           : number of clusters to find
 *     int iters       : number of clustering iterations to run
 *     int block_cnt   : number of blocks to use
 *     int threads_per_block: number of threads per block
 *
 * Output parameters:   
 *     struct point u[]: array of cluster centers
 *     int c[]         : cluster id for each data points
 */
void k_means(struct point p[MAX_POINTS], 
	     int m, 
	     int k,
	     int iters,
	     struct point u[MAX_CENTERS],
	     int c[MAX_POINTS],
	     int block_cnt,
	     int threads_per_block)
{
	int j;
	
	/* randomly initialized the centers */
	for(j = 0; j < k; j++)
		u[j] = random_center(p);
	/*
	 * To students: This function is not a GPU kernel. You need to add your
	 * own kernel functions in this file. Block count and thread count per
	 * count are provided as parameters to this functions. Please use these
	 * two parameters when launching your kernels.
	 */
        int *d_m, *d_k;
        int *d_c;
        struct point *d_u, *d_p;
        hipSetDevice(1);

        hipMalloc((void**)&d_m, sizeof(int));
        hipMalloc((void**)&d_k, sizeof(int));
        hipMalloc((void**)&d_p, MAX_POINTS*sizeof(struct point));

        hipMalloc((void**)&d_u, MAX_CENTERS*sizeof(struct point));
        hipMalloc((void**)&d_c, MAX_POINTS*sizeof(int));

        hipMemcpy(d_m, &m, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_k, &k, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_p, p, MAX_POINTS*sizeof(struct point), hipMemcpyHostToDevice);
        int l;
        for(l=0; l< iters; l++){
                hipMemcpy(d_c, c, MAX_POINTS*sizeof(int), hipMemcpyHostToDevice);
                hipMemcpy(d_u, u, MAX_CENTERS*sizeof(struct point), hipMemcpyHostToDevice);
                assignClusters<<<block_cnt, threads_per_block>>>(d_p, d_u, d_c, d_m, d_k, block_cnt, threads_per_block);
                //assignClusters(p, u, c, m, k);

                hipDeviceSynchronize();

                hipMemcpy(c, d_c, MAX_POINTS*sizeof(int), hipMemcpyDeviceToHost);
                computeNewCenter(p, u, c, m, k);
                //computeNewCenter<<<1,1>>>(d_p, d_u, d_c, d_m, d_k);
        }
        /*based on the cluster asignment, update the center for each cluster*/
        //hipMemcpy(u, d_u, MAX_CENTERS*sizeof(struct point), hipMemcpyDeviceToHost);
  	return;
}
